#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREADS_PER_BLOCK 32
#define TIME 3600000


__global__ void compute(float *a_d, float *b_d, float *c_d, float arraySize)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int t = threadIdx.x;
	int blockdim=blockDim.x;
		 
	if(ix<arraySize){
	if(ix==0){	
		b_d[ix]=200.0;
		}
		else{
		b_d[ix]=0.0;
		}
	}
	
	
	for(int k=0;k<TIME;k++) // time-loop
    {
	if( ix > 0 && ix < arraySize-1){
	   b_d[ix] = (b_d[ix+1]+b_d[ix-1])/2.0;
	}
	a_d[ix]=b_d[ix];
    	

}		
} 


extern "C" void pointsource_pollution (float *a, float *b, int *c, int arraySize)
{
	int numDevices = 0;    
	hipGetDeviceCount(&numDevices); 
	   if (numDevices > 1)
	    {       int maxMultiprocessors = 0, maxDevice = 0; 
	          for (int device=0; device<numDevices; device++) {          hipDeviceProp_t props;          hipGetDeviceProperties(&props, device); 
	                   if (maxMultiprocessors < props.multiProcessorCount) {           
	                     maxMultiprocessors = props.multiProcessorCount;  
	                                maxDevice = device;          }       }    
	       hipSetDevice(maxDevice);   } 

	float *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(float) * arraySize);
	hipMalloc ((void**) &b_d, sizeof(float) * arraySize);
	hipMalloc ((void**) &c_d, sizeof(float) * arraySize);
	

	compute <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);
	hipMemcpy (a, a_d, sizeof(float) * arraySize, hipMemcpyDeviceToHost);
	
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
		
	
}